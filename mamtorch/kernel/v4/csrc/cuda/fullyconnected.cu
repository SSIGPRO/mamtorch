#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <limits>

#define BSM 64 // block size along M
#define BSN BSM // block size along N
#define BSK 64 // block size along K
#define WPTM 4 // work per thread along M
#define WPTN WPTM // work per thread along N
#define RBSM (BSM/WPTM) // reduced block_size along M
#define RBSN (BSN/WPTN) // reduced block_size along N
#define LPTA ((BSK*BSM)/(RBSM*RBSN)) // loads-per-thread from global memory A
#define LPTB ((BSK*BSN)/(RBSM*RBSN)) // loads-per-thread from global memory B
#define LPTM BSM/RBSM
#define LPTK BSK/RBSN

/* OPTIMIZATION NOTES 
* - prefetch reduces performance due to the reduction of active thread for
*   each processor (avoided)
* - vectorization of data has not been tested
* - the use of transposition and padding introduce negligible delay
*/

namespace mamtorch_kernel_v4 {

__global__ void fullyconnected_cuda_kernel(
    const float * __restrict__ A,
    const float * __restrict__ BT,
    float * __restrict__ C,
    int * __restrict__ Cargmax,
    int * __restrict__ Cargmin,
    int M,
    int K,
    int N);

std::vector<at::Tensor> fullyconnected_cuda(
    at::Tensor A,
    at::Tensor B)
{   
    hipSetDevice(A.get_device()); // set GPU number
    
    // row-major to column-major + transpose
    const auto ATcm = A;
    // row-major to column-major + transpose
    const auto BTcm = B;

    // cuda matrices (A and B are swapped)
    auto Acuda = BTcm;
    auto Bcuda = ATcm;
    
    const auto M = Acuda.size(1);
    const auto K = Acuda.size(0);
    const auto N = Bcuda.size(0);
    
    auto BT = Bcuda.transpose(0,1).contiguous();

    // declare padded tensors
    at::Tensor A_padded = Acuda;
    at::Tensor BT_padded = BT;
    
    // evaluate padding to have matrix size multiple of BSM, BN, BSK
    int M_rest = M%BSM;
    int N_rest = N%BSN;
    int K_rest = K%BSK;
    int M_padding = 0;
    int N_padding = 0;
    int K_padding = 0;
    int M_padded = M;
    int N_padded = N;
    int K_padded = K;
    if(M_rest)
    {
        M_padding = BSM - M_rest;
        M_padded = M + M_padding;
    }
    if(N_rest)
    {
        N_padding = BSN - N_rest;
        N_padded = N + N_padding;
    }
    if(K_rest)
    {
        K_padding = BSK - K_rest;
        K_padded = K + K_padding;
    }
    
    // pad matrix A
    if(M_rest || K_rest)
    {
        A_padded = at::pad(Acuda.unsqueeze(0),
                           at::IntList{0, M_padding, 0, K_padding},
                           "replicate").squeeze();
    }
    
    // pad matrix BT
    if(N_rest || K_rest)
    {
        BT_padded = at::pad(BT.unsqueeze(0),
                            at::IntList{0, N_padding, 0, K_padding},
                            "replicate").squeeze();
    }
    
    // generate padded output matrix
    if(M_rest || N_rest)
    {
        
    }   
    auto C_padded = at::zeros({A.size(0), B.size(1)}, A.options());
    auto Cargmax_padded = at::zeros({A.size(0), B.size(1)}, A.options());
    auto Cargmin_padded = at::zeros({A.size(0), B.size(1)}, A.options());
    Cargmax_padded = Cargmax_padded.to(torch::kInt32);
    Cargmin_padded = Cargmin_padded.to(torch::kInt32);
    
    const dim3 threads(RBSM,
                       RBSN,
                       1);    
    const dim3 blocks(M_padded/BSM,
                      N_padded/BSN,
                      1);
    
    fullyconnected_cuda_kernel<<<blocks, threads>>>(
        A_padded.data_ptr<float>(),
        BT_padded.data_ptr<float>(),
        C_padded.data_ptr<float>(),
        Cargmax_padded.data_ptr<int>(),
        Cargmin_padded.data_ptr<int>(),
        M_padded, K_padded, N_padded);

    if(M_rest || N_rest)
    {
        C_padded = C_padded.slice(0, 0, N).slice(1, 0, M);
        Cargmax_padded = Cargmax_padded.slice(0, 0, N).slice(1, 0, M);
        Cargmin_padded = Cargmin_padded.slice(0, 0, N).slice(1, 0, M);
    }

    // transposed column-major to row-major -> identity
    auto C = C_padded;
    auto Cargmax = torch::clamp(Cargmax_padded, 0, K-1);
    auto Cargmin = torch::clamp(Cargmin_padded, 0, K-1);
    // NOTE: clamping is fundamental for the approximated computing kernel
    // when the maximum/minimum value is the last one, since padding is
    // performed with "replicate" option

    return {C, Cargmax, Cargmin};
}

} // end namespace mamtorch_kernel