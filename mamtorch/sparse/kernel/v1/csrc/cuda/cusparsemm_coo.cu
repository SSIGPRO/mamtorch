#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include <vector>
#include <cstring>

#include "cusparse_handle.h"

namespace mamtorch_kernel_sparsev1 {

static CUSPARSEHandle cusparse_handle;

using namespace torch::indexing;

torch::Tensor cusparsemm_coo(
    torch::Tensor A,
    torch::Tensor B)
{   
    hipSetDevice(A.get_device()); // set GPU number
    
    const auto N = A.size(0);
    const auto K = A.size(1);
    const auto M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    hipsparseDnMatDescr_t    Bdense, Cdense;
    hipsparseSpMatDescr_t    Asparse;
    void*                   dBuffer = NULL;
    size_t                  bufferSize = 0;

    // Create sparse matrix A in COO format
    hipsparseCreateCoo(&Asparse, N, K, A._nnz(), A.indices().data_ptr<int64_t>(), A.indices().data_ptr<int64_t>()+A._nnz(), A.values().data_ptr<float>(),
                    HIPSPARSE_INDEX_64I,
                    HIPSPARSE_INDEX_BASE_ZERO,
                    HIP_R_32F);

    // Convert dense matrices to cusparse format
    hipsparseCreateDnMat(&Bdense, K, M, M, B.data_ptr<float>(), HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&Cdense, N, M, M, C.data_ptr<float>(), HIP_R_32F, HIPSPARSE_ORDER_ROW);

    auto algorithm = HIPSPARSE_SPMM_COO_ALG4;

    // allocate external buffer if needed
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseSpMM_bufferSize(
        cusparse_handle.getHandle(),
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Asparse, Bdense, &beta, Cdense, HIP_R_32F,
        algorithm, &bufferSize
    );

    hipMalloc(&dBuffer, bufferSize);

    // perform sparse matmul
    hipsparseSpMM(
        cusparse_handle.getHandle(),
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Asparse, Bdense, &beta, Cdense, HIP_R_32F,
        algorithm, dBuffer
    );

    // destroy sparse handlers
    hipsparseDestroySpMat(Asparse);
    hipsparseDestroyDnMat(Bdense);
    hipsparseDestroyDnMat(Cdense);

    return C;
}

} // end namespace mamtorch